//2d cahn hillard with initial condition as random noise using spectral with periodic boundary conditions
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <hipfft/hipfft.h>


#define sizex 512
#define sizey 512
#define dt 1.0e-3
#define PI 3.14159265358979323846
#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32
#define NT 100000 //number of time steps

int iDivUp(int a, int b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

__global__ void iterate1(hipfftComplex *c, hipfftComplex *g)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	int p = i + j * sizex;

	g[p].x = 2 * c[p].x*(1 - c[p].x)*(1 - 2 * c[p].x);
}

__global__ void iterate2(hipfftComplex *c, hipfftComplex *g, double *dkx, double *dky)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	double kx, ky;
	if (i < sizex / 2)
		kx = i * *dkx;
	else
		kx = (i - sizex) * *dkx;

	if (j < sizey / 2)
		ky = j * *dky;
	else
		ky = (j - sizey) * *dky;

	int p = i + j * sizex;

	c[p].y = 0;
	g[p].y = 0;

	c[p].x = (c[p].x - dt * (kx*kx + ky * ky)*g[p].x) / (1 + 2 * (kx*kx + ky*ky) * (kx*kx + ky*ky) * dt);
}

__global__ void iterate3(hipfftComplex *c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	int p = i + j * sizex;

	c[p].x = c[p].x/(sizex*sizey);
	c[p].y = 0;
}

int main()
{
	clock_t t;
	t = clock();

	hipfftHandle plan;

	hipfftComplex c[sizex][sizey], *gpu_c, *gpu_g;

	char output_filename[100];
	char str1[]=".//output//order_parameter_";
	char str2[10];

	double *dkx, *dky;
	double *gpu_dkx, *gpu_dky;

	double dkx0 = 2 * PI / sizex;
	double dky0 = 2 * PI / sizey;

	dkx = &dkx0;
	dky = &dky0;

	srand(time(0));

	double random;

	FILE *file0;
	file0 = fopen(".//output//order_parameter_000.vtk", "w");

	if (file0 == NULL)
	{
		printf("Can't open order_parameter_000.vtk file for writting\n");
	}

	fprintf(file0, "# vtk DataFile Version 3.0\n");
	fprintf(file0, "Order Parameter data\n");
	fprintf(file0, "ASCII\n");
	fprintf(file0, "DATASET STRUCTURED_POINTS\n");
	fprintf(file0, "DIMENSIONS %d %d 1\n", sizex, sizey);
	fprintf(file0, "ORIGIN 0 0 0\n");
	fprintf(file0, "SPACING 1 1 1\n");
	fprintf(file0, "POINT_DATA %d\n", sizex*sizey);
	fprintf(file0, "SCALARS order_parameter double\n");
	fprintf(file0, "LOOKUP_TABLE default\n");

	for (int i = 0; i < sizex; i++)
	{
		for (int j = 0; j < sizey; j++)
		{
			random = rand();
			random /= RAND_MAX;
			random = 0.25 + 0.5*  random;
			c[i][j].x = random;
			c[i][j].y = 0.0f;

			fprintf(file0, "%f\n", c[i][j].x);
		}
	}

	fclose(file0);

	hipMalloc(&gpu_c, sizeof(hipfftComplex)*sizex*sizey);//note that gpu_c is 1D variable
	hipMalloc(&gpu_g, sizeof(hipfftComplex)*sizex*sizey);
	hipMalloc(&gpu_dkx, sizeof(double));
	hipMalloc(&gpu_dky, sizeof(double));

	hipMemcpy(gpu_c, c, sizeof(hipfftComplex)*sizex*sizey, hipMemcpyHostToDevice);
	hipMemcpy(gpu_dkx, dkx, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(gpu_dky, dky, sizeof(double), hipMemcpyHostToDevice);

	hipfftPlan2d(&plan, sizex, sizey, HIPFFT_C2C);

	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	dim3 dimGrid(iDivUp(sizex, BLOCK_SIZE_X), iDivUp(sizey, BLOCK_SIZE_Y));

	for (int t = 1; t <= NT; t++)
	{
		iterate1<<< dimGrid, dimBlock >>>(gpu_c, gpu_g);

		hipfftExecC2C(plan, gpu_c, gpu_c, HIPFFT_FORWARD);
		hipfftExecC2C(plan, gpu_g, gpu_g, HIPFFT_FORWARD);

		iterate2<<< dimGrid, dimBlock >>>(gpu_c, gpu_g, gpu_dkx, gpu_dky);

		hipfftExecC2C(plan, gpu_c, gpu_c, HIPFFT_BACKWARD);

		iterate3<<< dimGrid, dimBlock >>>(gpu_c);

		if(t%100 == 0)
		{		//print order parameter in vtk file
			hipMemcpy(c, gpu_c, sizeof(hipfftComplex)*sizex*sizey, hipMemcpyDeviceToHost);

			sprintf(str2, "%03d.vtk", t);
			strcat(output_filename, str1);
			strcat(output_filename, str2);

			FILE *file0;
			file0 = fopen(output_filename, "w");

			if (file0 == NULL)
			{
				printf("Can't open order_parameter_%03d.vtk file for writting\n", t);
			}

			fprintf(file0, "# vtk DataFile Version 3.0\n");
			fprintf(file0, "Order Parameter data\n");
			fprintf(file0, "ASCII\n");
			fprintf(file0, "DATASET STRUCTURED_POINTS\n");
			fprintf(file0, "DIMENSIONS %d %d 1\n", sizex, sizey);
			fprintf(file0, "ORIGIN 0 0 0\n");
			fprintf(file0, "SPACING 1 1 1\n");
			fprintf(file0, "POINT_DATA %d\n", sizex*sizey);
			fprintf(file0, "SCALARS order_parameter double\n");
			fprintf(file0, "LOOKUP_TABLE default\n");
	
			for (int x = 0; x < sizex; x++)
			{
				for (int y = 0; y < sizey; y++)
				{
					fprintf(file0, "%lf\n", c[x][y].x);
				}
			}

			fclose(file0);
			//printf("closed order_parameter_%03d.vtk file\n", t);

			for(int k=0;k<strlen(output_filename);k++)
			{
    				output_filename[k] = 0;
			}
		}
		printf("%f%%\n", (double)t*100/NT);

	}

	hipfftDestroy(plan);

	hipFree(gpu_c);
	hipFree(gpu_g);
	hipFree(gpu_dkx);
	hipFree(gpu_dky);

	t = clock() - t;
	double timetaken = ((double)t) / CLOCKS_PER_SEC;
	printf("time taken =%f\n", timetaken);

	return 0;
}
